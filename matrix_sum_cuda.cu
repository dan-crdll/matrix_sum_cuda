#include "hip/hip_runtime.h"
#include <iostream>
#include "matrix_utils.h"

__global__ void sum_matrix(int* matrix_1, int* matrix_2, int* result) {
    int idx = threadIdx.x;
    result[idx] = matrix_1[idx] + matrix_2[idx];
}


int main(int argc, char** argv) {
    int dimension;
    
    std::cout << "Insert matrix dimension : ";
    std::cin >> dimension;
    std::cout << std::endl;

    int* matrix = new int[dimension * dimension];
    int* matrix_2 = new int[dimension * dimension];
    int* result = new int[dimension * dimension];

    int* matrix_1_gpu;
    int* matrix_2_gpu;
    int* result_gpu;

    std::cout << "Matrix 1: " << std::endl;
    ask_values(dimension, matrix);
    std::cout << "Matrix 2: " << std::endl;
    ask_values(dimension, matrix_2);

    print_matrix(dimension, matrix);
    std::cout << " + " << std::endl;
    print_matrix(dimension, matrix_2);
    std::cout << " = " << std::endl;
    
    hipMalloc((void **)&matrix_1_gpu, dimension * dimension * sizeof(int));
    hipMalloc((void **)&matrix_2_gpu, dimension * dimension * sizeof(int));
    hipMalloc((void **)&result_gpu, dimension * dimension * sizeof(int));

    hipMemcpy(matrix_1_gpu, matrix, dimension * dimension * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(matrix_2_gpu, matrix_2, dimension * dimension * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(result_gpu, result, dimension * dimension * sizeof(int), hipMemcpyHostToDevice);

    sum_matrix<<<1, dimension * dimension>>>(matrix_1_gpu, matrix_2_gpu, result_gpu);
    hipMemcpy(result, result_gpu, dimension * dimension * sizeof(int), hipMemcpyDeviceToHost);

    print_matrix(dimension, result);

    hipFree(matrix_1_gpu);
    hipFree(matrix_2_gpu);
    hipFree(result_gpu);

    delete[] matrix;
    delete[] matrix_2;
    delete[] result;
}